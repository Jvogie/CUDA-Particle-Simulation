#include "hip/hip_runtime.h"
﻿#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>

struct Particle {
    float x, y; // Position
    float vx, vy; // Velocity
};

__global__ void update_particles(Particle* particles, int n, float deltaTime) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        particles[idx].x += particles[idx].vx * deltaTime;
        particles[idx].y += particles[idx].vy * deltaTime;
    }
}

void init_particles(Particle* particles, int n) {
    for (int i = 0; i < n; ++i) {
        particles[i].x = rand() % 1000 / 100.0f;
        particles[i].y = rand() % 1000 / 100.0f;
        particles[i].vx = rand() % 200 / 100.0f - 1.0f; // Velocity between -1.0 and 1.0
        particles[i].vy = rand() % 200 / 100.0f - 1.0f; // Velocity between -1.0 and 1.0
    }
}

int main() {
    int n = 1024; // Number of particles
    float deltaTime = 0.1f; // Time step

    Particle* particles = new Particle[n];
    Particle* d_particles;
    size_t size = n * sizeof(Particle);

    init_particles(particles, n);

    hipMalloc(&d_particles, size);
    hipMemcpy(d_particles, particles, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((n + threadsPerBlock.x - 1) / threadsPerBlock.x);

    update_particles << <blocksPerGrid, threadsPerBlock >> > (d_particles, n, deltaTime);

    hipMemcpy(particles, d_particles, size, hipMemcpyDeviceToHost);

    std::ofstream outFile("particle_positions.txt");
    for (int i = 0; i < n; i++) {
        outFile << particles[i].x << " " << particles[i].y << std::endl;
    }
    outFile.close();

    hipFree(d_particles);
    delete[] particles;
    return 0;
}
